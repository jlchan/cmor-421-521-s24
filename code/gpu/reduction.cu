#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 256

__global__ void partial_reduction(const int N, float *x_reduced, const float *x){
  
  __shared__ float s_x[BLOCKSIZE];

  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int tid = threadIdx.x;
  
  // coalesced reads in
  s_x[tid] = 0.f;
  if (i < N){
    s_x[tid] = x[i];
  }

  // number of "live" threads per block
  int alive = blockDim.x;
  
  while (alive > 1){
    __syncthreads(); 
    alive /= 2; // update the number of live threads    
    if (tid < alive){
      s_x[tid] += s_x[tid + alive];
    }
  }

  // write out once we're done reducing each block
  if (tid==0){
    x_reduced[blockIdx.x] = s_x[0];
  }
}
    
int main(int argc, char * argv[]){

  int N = 4096;
  if (argc > 1){
    N = atoi(argv[1]);
  }

  int blockSize = BLOCKSIZE;

  // Next largest multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize;

  printf("N = %d, blockSize = %d, numBlocks = %d\n", N, blockSize, numBlocks);

  float * x = new float[N];
  float * x_reduced = new float[numBlocks];  

  for (int i = 0; i < N; ++i){
    x[i] = 1.f;
  }

  // allocate memory and copy to the GPU
  float * d_x;
  float * d_x_reduced;  
  int size_x = N * sizeof(float);
  int size_x_reduced = numBlocks * sizeof(float);
  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_x_reduced, size_x_reduced);
  
  // copy memory over to the GPU
  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_x_reduced, x_reduced, size_x_reduced, hipMemcpyHostToDevice);

  partial_reduction <<< numBlocks, blockSize >>> (N, d_x_reduced, d_x);

  // copy memory back to the CPU
  hipMemcpy(x_reduced, d_x_reduced, size_x_reduced, hipMemcpyDeviceToHost);

  float sum_x = 0.f;
  for (int i = 0; i < numBlocks; ++i){
    sum_x += x_reduced[i];
  }

  //  float target = N * (N+1) / 2.f;
  float target = N;
  printf("error = %f\n", fabs(sum_x - target));

#if 1
  int num_trials = 10;
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < num_trials; ++i){
    partial_reduction <<< numBlocks, blockSize >>> (N, d_x_reduced, d_x);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  
  printf("Time to run kernel 10x: %6.2f ms.\n", time);
  
#endif

  return 0;
}
