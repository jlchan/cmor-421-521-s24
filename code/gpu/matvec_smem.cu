#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define NCOLS 64

__global__ void matvec(int N, const float *A, float *x, float *y){
  
  __shared__ float s_x[NCOLS];
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  int ii = threadIdx.x;
  while (ii < NCOLS){
    s_x[ii] = x[ii];
    ii += blockDim.x;
  }
  
  __syncthreads();
  
  if (i < N){
    float val = y[i];
    for (int j = 0; j < NCOLS; ++j){
      val += A[i + j * N] * s_x[j];  
      //val += A[i + j * N] * x[j];  
    }
    y[i] = val;
  }
}
    
int main(int argc, char * argv[]){

  int N = 4096;
  int blockSize = 32;
  if (argc > 1){
    N = atoi(argv[1]);
    blockSize = atoi(argv[2]);
  }
  printf("N = %d, blockSize = %d\n", N, blockSize);

  float * A = new float[N * NCOLS];
  float * x = new float[NCOLS];
  float * y = new float[N];

  for (int i = 0; i < N; ++i){
    for (int j = 0; j < NCOLS; ++j){
      A[i + j * N] = 1.f / NCOLS;
    }
    if (i < NCOLS){
      x[i] = 1.f;
    }
    y[i] = 0.f;
  }

  // allocate memory and copy to the GPU
  float * d_A;
  float * d_x;
  float * d_y;
  int size_A = N * NCOLS * sizeof(float);
  int size_x = NCOLS * sizeof(float);
  int size_y = N * sizeof(float);
  hipMalloc((void **) &d_A, size_A);
  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_y);
  
  // copy memory over to the GPU
  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_y, hipMemcpyHostToDevice);

  // Next largest multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize; 
  matvec <<< numBlocks, blockSize >>> (N, d_A, d_x, d_y);

  // copy memory back to the CPU
  hipMemcpy(y, d_y, size_y, hipMemcpyDeviceToHost);
  
  float error = 0.f;
  for (int i = 0; i < N; ++i){
    error += fabs(y[i] - 1.f);
  }
  printf("error = %f\n", error);

#if 1
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  for (int i = 0; i < 10; ++i){
    matvec <<< numBlocks, blockSize >>> (N, d_A, d_x, d_y);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  
  printf("Time to run kernel: %6.2f ms \n", time);
  
#endif

  return 0;
}
