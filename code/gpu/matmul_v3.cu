#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 32

__global__ void matmul(int N, const float *A, const float *B, float *C) {
  
  // the output block that we want to compute in this threadblock
  const int cRow = blockIdx.x;
  const int cCol = blockIdx.y;

  // allocate buffer for current block in fast shared mem
  // shared mem is shared between all threads in a block
  __shared__ float s_A[BLOCKSIZE * BLOCKSIZE];
  __shared__ float s_B[BLOCKSIZE * BLOCKSIZE];

  // the inner row & col that we're accessing in this thread
  const int threadCol = threadIdx.x % BLOCKSIZE;
  const int threadRow = threadIdx.x / BLOCKSIZE;

  // advance pointers to the starting positions
  A += cRow * BLOCKSIZE * N;                    // row=cRow, col=0
  B += cCol * BLOCKSIZE;                        // row=0, col=cCol
  C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

  float tmp = 0.0;
  for (int blkIdx = 0; blkIdx < N; blkIdx += BLOCKSIZE) {
    // Have each thread load one of the elements in A & B
    // Make the threadCol (=threadIdx.x) the consecutive index
    // to allow global memory access coalescing
    s_A[threadRow * BLOCKSIZE + threadCol] = A[threadRow * N + threadCol];
    s_B[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];

    // block threads in this block until cache is fully populated
    __syncthreads();

    A += BLOCKSIZE;
    B += BLOCKSIZE * N;

    // execute the dotproduct on the currently cached block
    for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
      tmp += s_A[threadRow * BLOCKSIZE + dotIdx] * s_B[dotIdx * BLOCKSIZE + threadCol];
    }
    // need to sync again at the end, to avoid faster threads
    // fetching the next block into the cache before slower threads are done
    __syncthreads();
  }
  C[threadRow * N + threadCol] += tmp;
}

int main(int argc, char * argv[]){

  int N = 4096;
  if (argc > 1){
    N = atoi(argv[1]);
  }

  float * A = new float[N * N];
  float * B = new float[N * N];
  float * C = new float[N * N];

  for (int i = 0; i < N * N; ++i){
    A[i] = 0.f;
    B[i] = 0.f;
    C[i] = 0.f;
  }
  for (int i = 0; i < N; ++i){
    A[i + i * N] = 1.f; // identity
    B[i + i * N] = 1.f; // identity
  }

  // allocate memory and copy to the GPU
  float * d_A;
  float * d_B;
  float * d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);
  
  // copy memory over to the GPU
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

  // Next largest multiple of blockSize
  int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE; 
  printf("N = %d, numBlocks * blockSize = %d\n", N, numBlocks * BLOCKSIZE);
  dim3 gridDims(numBlocks, numBlocks);
  dim3 blockDims(BLOCKSIZE * BLOCKSIZE);
  matmul <<< gridDims, blockDims >>> (N, d_A, d_B, d_C);

  // copy memory back to the CPU
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  
  float error = 0.f;
  for (int i = 0; i < N; ++i){
    for (int j = 0; j < N; ++j){
      //      printf("C[%d,%d] = %f\n", i, j, C[j + i * N]);
      float Cij = 0.f;
      if (i==j){
	Cij = 1.f;
      }
      float diff = C[j + i * N] - Cij;
      error += fabs(diff);
    }
  }
  printf("error = %f\n", error);


  return 0;
}
